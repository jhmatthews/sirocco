/* ****************************************************************************************************************** */
/**
 *  @file matrix_gpu.cu
 *  @author Edward J. Parkinson (e.parkinson@soton.ac.uk)
 *  @date August 2023
 *
 *  @brief
 *
 *  ***************************************************************************************************************** */


#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>

#if CUDA_ON

#include <cuda.h>
#include <cuda_runtime.h>
#include <cusolverDn.h>

/* NVCC is a C++ compiler at heart, so anything we re-use from regular C source
   has to be defined here with `extern "C"` to tell the compiler that the
   function has been compiled by a C compiler (and does some computer science
   stuff to make linking possible) */

extern "C" int Exit (int error_code);
extern "C" int Error (const char *format, ...);
extern "C" int Log (const char *format, ...);

/* `cusolver_handle` is a variable used to interact with the cuSolver/CUDA
    runtime and is used to initialise and clean up the resources required for
    both runtimes */

static cusolverDnHandle_t cusolver_handle = NULL;

/* ****************************************************************************************************************** */
/**
 *  @brief Check the return status of a CUDA function
 *
 *  @param [in] status  the status to check
 *
 *  ***************************************************************************************************************** */

#define CUDA_CHECK(status)                                                                                             \
  do {                                                                                                                 \
    cudaError_t err = status;                                                                                          \
    if (err != cudaSuccess) {                                                                                          \
      Error("CUDA Error: %s\n", cudaGetErrorString(err));                                                              \
      Exit(EXIT_FAILURE);                                                                                              \
    }                                                                                                                  \
  } while (0)

/* ****************************************************************************************************************** */
/**
 *  @brief Check the return status of a cuSOLVER function
 *
 *  @param [in] status  the status to check
 *
 *  ***************************************************************************************************************** */

#define CUSOLVER_CHECK(status)                                                                                         \
  do {                                                                                                                 \
    cusolverStatus_t err = status;                                                                                     \
    if (err != CUSOLVER_STATUS_SUCCESS) {                                                                              \
      Error("cuSolver Error: %d\n", err);                                                                              \
      Exit(EXIT_FAILURE);                                                                                              \
    }                                                                                                                  \
  } while (0)

/* ****************************************************************************************************************** */
/**
 * @brief
 *
 * @return
 *
 * @details
 *
 *  ***************************************************************************************************************** */

extern "C" void
create_cusolver_handle (void)
{
  CUSOLVER_CHECK (cusolverDnCreate (&cusolver_handle));
  Log ("Created a new cuSOLVER handle created\n");
}

/* ****************************************************************************************************************** */
/**
 * @brief
 *
 * @return
 *
 * @details
 *
 *  ***************************************************************************************************************** */

extern "C" void
destroy_cusolver_handle (void)
{
  CUSOLVER_CHECK (cusolverDnDestroy (cusolver_handle));
  Log ("Destroyed the cuSOLVER handle\n");
}

/* ****************************************************************************************************************** */
/**
 * @brief  Solve the linear system A x = b, for the vector x
 *
 * @param  [in]  a_matrix  a square matrix on the LHS
 * @param  [in]  b_vector  the B resultant vector
 * @param  [in]  size  the number of rows (and columns) in the square matrix matrix and vectors
 * @param  [out] x_vector  the x vector on the RHS
 *
 * @return an integer representing the error state
 *
 * @details
 * Performs LU decomposition to solve for x in the linear system A x = b. The calculation is perform, in serial, on
 * the CPU using GSL.
 *
 *  ***************************************************************************************************************** */

extern "C" int
gpu_solve_linear_system (double *a_matrix, double *b_vector, int size, double *x_vector)
{
  if (cusolver_handle == NULL)
    create_cusolver_handle ();

  int *devInfo;
  int lwork;
  int *d_pivot;                 /* device array of pivoting sequence */
  double *d_A, *d_b;
  double *d_work;               /* cuSolver needs a "workspace" to do stuff, which we have to allocate manually */

  /* Allocate memory on the GPU (device) to store the matrices/vectors */
  cudaMalloc ((void **) &d_A, size * size * sizeof (double));
  cudaMalloc ((void **) &d_b, size * sizeof (double));
  cudaMalloc ((void **) &devInfo, sizeof (int));

  /* Copy the matrix and vector to the device memory */
  cudaMemcpy (d_A, a_matrix, size * size * sizeof (double), cudaMemcpyHostToDevice);
  cudaMemcpy (d_b, b_vector, size * sizeof (double), cudaMemcpyHostToDevice);
  cudaMalloc ((void **) &d_pivot, size * sizeof (int));

  /* XXXX_bufferSize is used to compute the size of the workspace we need, and depends on the size of the linear
     system being solved */
  cusolverDnDgetrf_bufferSize (cusolver_handle, size, size, d_A, size, &lwork);
  cudaMalloc ((void **) &d_work, lwork * sizeof (double));

  /* Perform LU factorization and solve the linear system. The vector d_b is not used in `getrs` (the solver), but
     it's the same size of the solution vector so we'll re-use that. d_b is then copied back to host memory (CPU RAM) */
  cusolverDnDgetrf (cusolver_handle, size, size, d_A, size, d_work, d_pivot, devInfo);
  cusolverDnDgetrs (cusolver_handle, CUBLAS_OP_T, size, 1, d_A, size, d_pivot, d_b, size, devInfo);
  cudaMemcpy (x_vector, d_b, size * sizeof (double), cudaMemcpyDeviceToHost);

  cudaFree (d_A);
  cudaFree (d_b);
  cudaFree (d_work);
  cudaFree (d_pivot);

  return EXIT_SUCCESS;
}

/* ****************************************************************************************************************** */
/**
 * @brief
 *
 * @param  [in]  a_matrix
 * @param  [out] a_inverse
 * @param  [in]  num_rows
 *
 * @return an integer representing the error state
 *
 * @details
 *
 *  ***************************************************************************************************************** */

extern "C" int
gpu_invert_matrix (double *matrix, double *inverse, int num_rows)
{
  if (!cusolver_handle)
    create_cusolver_handle ();

  return EXIT_SUCCESS;
}

#endif
